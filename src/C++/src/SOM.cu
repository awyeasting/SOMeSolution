#include "hip/hip_runtime.h"
/*
 * This file is part of SOMeSolution.
 *
 * Developed for Pacific Northwest National Laboratory.
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the BSD 3-Clause License as published by
 * the Software Package Data Exchange.
 */

#include "SOM.h"

//----------------------------------------------------
//	CUDA KERNEL FUNCTIONS
//----------------------------------------------------

/*
	CUDA kernel function for performing elementwise multiplication on two matrices</summary>
*/
__global__
void elementMul(double *A, double *B, double *C, int n) {
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (i < n)
		C[i] = A[i] * B[i];
}

/*
	CUDA kernel function for filling a matrix with ones
*/
__global__
void fillOnes(double *A, int n) {
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (i < n)
		A[i] = 1.0f;
}

/*
	CUDA kernel function calculating the BMUs of nodes as found by distances in the D matrix
*/
__global__
void findBMUsGPU(double *D, int *BMUs, int xdn, int nnodes) {
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (i < xdn) {
		// TODO: Optimize this further to utilize more processors
		BMUs[i] = 0;
		for (int j = 1; j < nnodes; j++) {
			// Uses column major order
			if (D[BMUs[i] * xdn + i] > D[j * xdn + i])
				BMUs[i] = j;
		}
	}
}

/*
	CUDA kernel function for calculating the gaussian value as described in the paper by Liu et. al.
*/
__global__
void calcGaussian(double *H, int xdn, int nnodes, double initial_map_radius, double neighborhood_radius, int *BMUs, int height) {
	int row = (blockIdx.y * blockDim.y) + threadIdx.y;
	int col = (blockIdx.x * blockDim.x) + threadIdx.x;
	
	if ((row < xdn) && (col < nnodes)) {
		int col_y = col % height;
		int col_x = (col - col_y) / height;

		// Get BMU coord
		int row_y = BMUs[row] % height;
		int row_x = (BMUs[row] - row_y) / height;

		H[col*xdn + row] = initial_map_radius * exp(-(double)((row_x - col_x) * (row_x - col_x) + (row_y - col_y) * (row_y - col_y))/(neighborhood_radius * neighborhood_radius));
	}
}

/*
	CUDA kernel function for copying a matrix from row major order to column major order
*/
__global__
void rowToColumnMajor(double *idata, double *odata, int nrows, int ncols, int n) {
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (i < n)
		odata[(i%ncols)*nrows + (i/ncols)] = idata[i];
}

//----------------------------------------------------
//	SOM non-member functions
//----------------------------------------------------

void trainOneEpoch(hipblasHandle_t &handle, int device, double *train, double *weights, double *numer, double *denom, int map_size, int height, int num_examples, int dimensions, double initial_map_radius, double neighborhood_radius) {

	// Set assigned gpu
	gpuErrchk(hipSetDevice(device));

	// Find BMUs for every input instance
	// D = X_sq - 2X^TM + M_sq
	// D (xdn * nn)
	
	// Calc m_sq
	// Elementwise multiply M by M
	double *d_msq;
	gpuErrchk(hipMalloc(&d_msq, map_size * dimensions * sizeof(double)));
	int NUM_THREADS = 256;
	int NUM_BLOCKS = (int) ceil((float)(map_size*dimensions)/NUM_THREADS);
	elementMul<<<NUM_BLOCKS, NUM_THREADS>>>(weights, weights, d_msq, map_size * dimensions);
	// Left multiply elementwise multiplied M by all ones matrix (of dim num examples x dimensions)
	double *d_o;
	gpuErrchk(hipMalloc(&d_o, num_examples * dimensions * sizeof(double)));
	NUM_BLOCKS = (int) ceil((float)(num_examples * dimensions)/NUM_THREADS);
	fillOnes<<<NUM_BLOCKS,NUM_THREADS>>>(d_o, num_examples * dimensions);
	gpuErrchk(hipDeviceSynchronize());
	// m_sq = ones x (M * M)^T
	const double alpha0 = 1.0f;
	const double beta0 = 0.0f;
	double *m_sq;
	gpuErrchk(hipMalloc(&m_sq, num_examples * map_size * sizeof(double)));
	hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, num_examples, map_size, dimensions, &alpha0, d_o, num_examples, d_msq, map_size, &beta0, m_sq, num_examples);
	
	gpuErrchk(hipDeviceSynchronize());
	
	gpuErrchk(hipFree(d_msq));
	gpuErrchk(hipFree(d_o));

	// Calc x_sq
	// Elementwise multiply X by X
	double *d_xsq;
	gpuErrchk(hipMalloc(&d_xsq, num_examples * dimensions * sizeof(double)));
	NUM_BLOCKS = (int) ceil((float)(num_examples*dimensions)/NUM_THREADS);
	elementMul<<<NUM_BLOCKS, NUM_THREADS>>>(train, train, d_xsq, num_examples * dimensions);
	gpuErrchk(hipDeviceSynchronize());
	// Left multiply elementwise multiplied X by all ones matrix (of dim num examples x dimensions)
	gpuErrchk(hipMalloc(&d_o, dimensions * map_size * sizeof(double)));
	NUM_BLOCKS = (int) ceil((float)(dimensions * map_size)/NUM_THREADS);
	fillOnes<<<NUM_BLOCKS,NUM_THREADS>>>(d_o, dimensions * map_size);
	// x_sq = (X * X) x ones
	double *x_sq;
	gpuErrchk(hipMalloc(&x_sq, num_examples * map_size * sizeof(double)));
	hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, num_examples, map_size, dimensions, &alpha0, d_xsq, num_examples, d_o, dimensions, &beta0, x_sq, num_examples);
	
	gpuErrchk(hipDeviceSynchronize());

	gpuErrchk(hipFree(d_xsq));
	gpuErrchk(hipFree(d_o));

	// Calc D
	// From paper: 
	// D = x_sq - 2 * x^t * m + m_sq

	const double alpha1 = -2.0f;
	const double beta1 = 1.0f;

	gpuErrchk(hipDeviceSynchronize());

	// m_sq = - 2 * (x^t * m) + (m_sq)
	hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, num_examples, map_size, dimensions, &alpha1, train, num_examples, weights, map_size, &beta1, m_sq, num_examples);

	gpuErrchk(hipDeviceSynchronize());

	// D = (x_sq) + (-2 * x^t * m + m_sq)
	double *D;
	gpuErrchk(hipMalloc(&D, num_examples * map_size * sizeof(double)));
	hipblasDgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, num_examples, map_size, &alpha0, x_sq, num_examples, &beta1, m_sq, num_examples, D, num_examples);

	gpuErrchk(hipDeviceSynchronize());

	gpuErrchk(hipFree(m_sq));
	gpuErrchk(hipFree(x_sq));

	// BMU index of each training instance
	int *BMUs;
	gpuErrchk(hipMalloc(&BMUs, num_examples * sizeof(int)));
	NUM_BLOCKS = (int) ceil((float)(num_examples)/NUM_THREADS);
	findBMUsGPU<<<NUM_BLOCKS, NUM_THREADS>>>(D, BMUs, num_examples, map_size);

	gpuErrchk(hipDeviceSynchronize());

	gpuErrchk(hipFree(D));

	// Calc gaussian function 
	// (num_examples x num nodes)
	int BLOCK_SIZE = 16;
	int GRID_HEIGHT = (int)ceil((float)num_examples/BLOCK_SIZE);
	int GRID_WIDTH = (int)ceil((float)map_size/BLOCK_SIZE);
	dim3 grid(GRID_WIDTH, GRID_HEIGHT);
	dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
	double *H;
	gpuErrchk(hipMalloc(&H, num_examples * map_size * sizeof(double)));
	calcGaussian<<<grid, threads>>>(H, num_examples, map_size, initial_map_radius, neighborhood_radius, BMUs, height);
	gpuErrchk(hipDeviceSynchronize());

	gpuErrchk(hipFree(BMUs));

	// Calc denominators
	// Left multiply H by a num_examples dimensional vector of ones
	gpuErrchk(hipMalloc(&d_o, num_examples * sizeof(double)));
	NUM_BLOCKS = (int)ceil((float)num_examples/NUM_THREADS);
	
	fillOnes<<<NUM_BLOCKS,NUM_THREADS>>>(d_o, num_examples);
	gpuErrchk(hipDeviceSynchronize());
	// denom = ones^T (1 x num examples) * H (num examples x map size)
	hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 1, map_size, num_examples, &alpha0, d_o, 1, H, num_examples, &beta0, denom, 1);
	
	gpuErrchk(hipDeviceSynchronize());
	
	gpuErrchk(hipFree(d_o));
	
	// Calc numerators
	// numer = H^T x X
	hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, map_size, dimensions, num_examples, &alpha0, H, num_examples, train, num_examples, &beta0, numer, map_size);

	gpuErrchk(hipDeviceSynchronize());
	gpuErrchk(hipFree(H));
}

//----------------------------------------------------
//	public SOM functions
//----------------------------------------------------

/* 
	Construct untrained SOM with given lattice width and height
*/
SOM::SOM(unsigned int width, unsigned int height)
{
	MPI_Comm_rank(MPI_COMM_WORLD, &this->_rank);
	MPI_Comm_size(MPI_COMM_WORLD, &this->_numProcs);
	//this->_rank = MPI::COMM_WORLD.Get_rank();
	//this->_numProcs = MPI::COMM_WORLD.Get_size();

	this->_width = width;
	this->_height = height;
}

/*
	Construct SOM from a saved SOM width, height, and set of weights
*/
SOM::SOM(std::istream &in) {
	MPI_Comm_rank(MPI_COMM_WORLD, &this->_rank);
	MPI_Comm_size(MPI_COMM_WORLD, &this->_numProcs);
	//this->_rank = MPI::COMM_WORLD.Get_rank();
	//this->_numProcs = MPI::COMM_WORLD.Get_size();

	this->loadWeights(in);
}

/*
	Generates a random set of training data if there is no input file given
*/
void SOM::gen_train_data(unsigned int num_examples, unsigned int dimensions, unsigned int seedValue)
{
	if (_trainData != NULL) {
		std::cout << "WARNING: train data not initialized" << std::endl;
	}
	this->_dimensions = dimensions;
	// TODO: Switch to compute based examples distribution
	this->_numExamples = num_examples / this->_numProcs;
	this->_trainData = new double [this->_numExamples * this->_dimensions];
	srand(seedValue + this->_rank);
	for (int i = 0; i < this->_numExamples; i++)
	{
		int rowMod = (this->_numExamples - i - 1) * this->_dimensions;
		for (int d = 0; d < this->_dimensions; d++)
		{
			double weight = SOM::randWeight();
			this->_trainData[rowMod + d] = weight;
		}
	}
}

/*
	Load a set of training data from a given filename

	Precondition: File is already open
*/
bool SOM::load_train_data(std::string fileName, bool hasLabelRow, bool hasLabelColumn) {
	unsigned int cols = 0, rows = 0;
	bool okOpen = true;
	if (this->_rank == 0) {
		// Open file for counting number of rows and columns
		std::ifstream infile(fileName, std::ifstream::in);
		if (!infile.is_open()) {
			okOpen = false;
			std::cout << "Invalid training data file '" << fileName << "'" << std::endl;
		} else {
			// Read in first row of data into line
			std::string line;
			if (hasLabelRow) {
				std::getline(infile, line);
				rows++;
			}
			std::getline(infile, line);
			rows++;

			// Count number of values in the first row to determine num columns
			// TODO: make this work with non number labels
			std::stringstream ss(line);
			double temp;
			while (ss >> temp) {
				cols++;
			}
			while(std::getline(infile, line)) {
				// Ignore empty lines to be more forgiving of minor formatting mistakes
				if (line.compare("") != 0)
					rows++;
			}
		}
		infile.close();
	}
	// Check there was no problem with the file
	MPI_Bcast(&okOpen, 1, MPI_C_BOOL, 0, MPI_COMM_WORLD);
	if (!okOpen) {
		return false;
	}

	// Broadcast rows and columns
	MPI_Bcast(&rows, 1, MPI_UNSIGNED, 0, MPI_COMM_WORLD);
	MPI_Bcast(&cols, 1, MPI_UNSIGNED, 0, MPI_COMM_WORLD);

	this->_numExamples = rows - ((unsigned int)hasLabelRow);
	this->_dimensions = cols - ((unsigned int)hasLabelColumn);
	
	// Initialize feature maximums and minimums
	this->_featureMaxes = (double *)malloc(sizeof(double) * this->_dimensions);
	this->_featureMins = (double*)malloc(sizeof(double) * this->_dimensions);
	for(int i =0; i < this->_dimensions; i++){
		_featureMaxes[i] = std::numeric_limits<double>::min();
		_featureMins[i] = std::numeric_limits<double>::max();
	}

	// Calculate starting position
	int read_count = this->_numExamples / this->_numProcs;
	int startRow = ((this->_numExamples / this->_numProcs) * this->_rank);
	// Adjust for remainder of examples
	if (this->_rank == this->_numProcs - 1) {
		read_count += (this->_numExamples % this->_numProcs);
	}
	this->_numExamples = read_count;

	// Prepare for reading in assigned chunk
	bool readOk = true;
	std::fstream infile(fileName, std::ifstream::in); // Assume because it opened for rank 0 it will open for all
	std::fstream& procfile = GotoLine(infile, startRow);
	this->_trainData = (double *)malloc(this->_numExamples * this->_dimensions * sizeof(double));

	// Read in assigned portion
	int procSectionLineNum = 0;
	std::string line;
	while(procSectionLineNum < read_count && std::getline(procfile, line)) {
		if (line.compare("") != 0) {
			std::stringstream ss(line);
			double temp;
			int cols_count = 0;
			// Read line into train data
			while (ss >> temp && cols_count < this->_dimensions) {
				this->_trainData[procSectionLineNum * this->_dimensions + cols_count] = temp;
				if (temp > this->_featureMaxes[cols_count]) {
					this->_featureMaxes[cols_count] = temp;
				}
				if (temp < this->_featureMins[cols_count]) {
					this->_featureMins[cols_count] = temp;
				}
				cols_count++;
			}
			// If the line finished reading early then the data is not of a consistent dimension
			if (cols_count != this->_dimensions - 1) {
				readOk = false;
			}
			procSectionLineNum++;
		}
		if (!readOk)
			break;
	}
	// If it didn't read enough lines then the data is not properly formatted
	if (procSectionLineNum != read_count - 1) {
		readOk = false;
		return false;
	}

	// Check that all the threads read their data properly
	bool allReadOk;
	MPI_Barrier(MPI_COMM_WORLD);
	MPI_Allreduce(&readOk, &allReadOk, 1, MPI_C_BOOL, MPI_LAND, MPI_COMM_WORLD);
	// If any process failed to read the process report it and discharge any allocated memory
	if (!allReadOk) {
		destroy_train_data();
		std::cout << "Error reading input file: Unable to read input file, check to make sure the input is properly formatted" << std::endl;
		return false;
	}

	// Find the true feature maxes and true feature mins
	double *globalMaxes = (double *)malloc(sizeof(double) * this->_dimensions);
	double *globalMins = (double*)malloc(sizeof(double) * this->_dimensions);
	MPI_Allreduce(this->_featureMaxes, globalMaxes, this->_dimensions, MPI_DOUBLE, MPI_MAX, MPI_COMM_WORLD);
	MPI_Allreduce(this->_featureMins, globalMins, this->_dimensions, MPI_DOUBLE, MPI_MIN, MPI_COMM_WORLD);
		
	// Free pre reduction maxes and mins
	free(this->_featureMaxes);
	free(this->_featureMins);
	this->_featureMaxes = globalMaxes;
	this->_featureMins = globalMins;

	return true;
}

void SOM::destroy_train_data() {
	free(this->_trainData);
	free(this->_featureMaxes);
	free(this->_featureMins);

	this->_trainData = NULL;
	this->_featureMaxes = NULL;
	this->_featureMins = NULL;
}

/*
	Train the SOM using a set of training data over a given number of epochs with a given learning rate
*/
void SOM::train_data(unsigned int epochs, double initial_learning_rate, unsigned int map_seed)
{
	// Check that the training data has been loaded in
	if (this->_trainData == NULL) {
		std::cout << "Train data not yet initialized in SOM" << std::endl;
		return;
	}

	this->_mapSize = this->_width * this->_height;

	hipblasHandle_t* handles;
	double neighborhood_radius, *numer, *denom, **d_train, **d_weights, **d_numer, **d_denom, **gnumer, **gdenom;
	int NUM_GPUS, *GPU_EXAMPLES, *GPU_OFFSET;

	// Establish multi gpu setup on current node
	// TODO: Add num gpus option
	hipGetDeviceCount(&NUM_GPUS);
	omp_set_dynamic(0); // Disable dynamic teams
	omp_set_num_threads(NUM_GPUS);

	// Allocate memory associated with training on each GPU on each node
	initNumDenom(numer, denom);
	initGPUTrainMemory(NUM_GPUS, handles, d_train, d_weights, d_numer, d_denom, GPU_EXAMPLES, GPU_OFFSET, this->_numExamples);
	initGPUNumDenReducMem(NUM_GPUS, gnumer, gdenom);
	normalizeData(this->_trainData);
	// Split training data onto gpus on each node
	initGPUTrainData(NUM_GPUS, this->_trainData, d_train, GPU_EXAMPLES, GPU_OFFSET);
	
	// TODO: verify that global_numer and denom only need to be allocated on rank 0
	double* global_numer;
	double* global_denom;
	// Init codebook on first node
	if (this->_rank == 0) {
		srand(map_seed);
		if (GPU_BASED_CODEBOOK_INIT)
			initCodebookOnGPU(d_weights);
		else
			initCodebook();

		// Init global numerators and denominators for reduction to node 0
		global_numer = (double*)malloc(_width * _height * _dimensions*sizeof(double));
		global_denom = (double *)malloc(_width * _height * sizeof(double));
	}

	double initial_map_radius = _width < _height ? ((double)_width) / 2.0 : ((double)_height) / 2.0;
	double time_constant = double(epochs) / log(initial_map_radius);
	
	for(int epoch = 0; epoch < epochs; epoch++) {
		// Wait for all other nodes to start the epoch
		MPI_Barrier(MPI_COMM_WORLD);

		// Send out the map on proc 0
		MPI_Bcast(this->_weights, this->_width * this->_height * this->_dimensions, MPI_DOUBLE, 0, MPI_COMM_WORLD);

		// Update gpu copies of the map
		setGPUCodebooks(d_weights);

		// Calculate current neighborhood radius
		neighborhood_radius = initial_map_radius * exp(-((double)(epoch))/time_constant);
		// Train a single epoch on all gpus
		#pragma omp parallel
		{
			int gpu = omp_get_thread_num();
			gpuErrchk(hipSetDevice(gpu));
			gpuErrchk(hipDeviceSynchronize());
			trainOneEpoch(handles[gpu], gpu, d_train[gpu], d_weights[gpu], d_numer[gpu], d_denom[gpu], this->_mapSize, this->_height, GPU_EXAMPLES[gpu], this->_dimensions, initial_map_radius, neighborhood_radius);
			gpuErrchk(hipMemcpy(gnumer[gpu],d_numer[gpu], this->_mapSize * this->_dimensions * sizeof(double), hipMemcpyDeviceToHost));
			gpuErrchk(hipMemcpy(gdenom[gpu],d_denom[gpu], this->_mapSize * sizeof(double), hipMemcpyDeviceToHost));
		}

		// Reduce numerators and denominators across gpus on proc
		// TODO: Implement more complex reduction
		for(int gpu = 0; gpu < NUM_GPUS; gpu++) {
			if (gpu == 0) {
				for (int i = 0; i < this->_mapSize; i++) {
					denom[i] = gdenom[gpu][i];
					for (int d = 0; d < this->_dimensions; d++) {
						numer[d*this->_mapSize + i] = gnumer[gpu][d*this->_mapSize + i];
					}
				}
			} else {
				for (int i = 0; i < this->_mapSize; i++) {
					denom[i] += gdenom[gpu][i];
					for (int d = 0; d < this->_dimensions; d++) {
						numer[d*this->_mapSize + i] += gnumer[gpu][d*this->_mapSize + i];
					}
				}
			}
		}

		// Reduce numerators and denominators across all procs
		MPI_Reduce(numer, global_numer, this->_mapSize * this->_dimensions, MPI_DOUBLE, MPI_SUM, 0, MPI_COMM_WORLD);
		MPI_Reduce(numer, global_numer, this->_mapSize, MPI_DOUBLE, MPI_SUM, 0, MPI_COMM_WORLD);
		
		// Update codebook/map
		if (this->_rank == 0) {
			// Recalculate weights with new numerators and denominators
			for (int i = 0; i < this->_mapSize; i++) {
				for (int d = 0; d < this->_dimensions; d++) {
					this->_weights[d*this->_mapSize + i] = numer[d*this->_mapSize + i] / denom[i];
				}
			}
		}
	}

	// Perform column major to row major order on weights matrix
	double *tempWeights = (double *)malloc(this->_mapSize * this->_dimensions * sizeof(double));
	for (int i = 0; i < this->_mapSize; i++) {
		for (int d = 0; d < this->_dimensions; d++) {
			tempWeights[i*this->_dimensions + d] = this->_weights[d*this->_mapSize + i];
		}
	}
	free(this->_weights);
	this->_weights = tempWeights;

	for (int gpu = 0; gpu < NUM_GPUS; gpu++) {
		hipSetDevice(gpu);
		hipblasDestroy(handles[gpu]);
		hipFree(d_train[gpu]);
		hipFree(d_weights[gpu]);
		hipFree(d_numer[gpu]);
		hipFree(d_denom[gpu]);
		free(gnumer[gpu]);
		free(gdenom[gpu]);
	}

	free(GPU_EXAMPLES);
	free(GPU_OFFSET);
	free(handles);
	free(d_train);
	free(d_weights);
	free(d_numer);
	free(d_denom);
	free(gnumer);
	free(gdenom);
	free(numer);
	free(denom);
	free(global_numer);
	free(global_denom);
}

/*
	Save the width and height of the SOM followed by the weights for each node with a different node's weights on every line
*/
void SOM::save_weights(std::ostream &out)
{
	out << this->_width << " " << this->_height << std::endl;
	for (int i = 0; i < this->_width; i++)
	{
		for (int j = 0; j < this->_height; j++)
		{
			for (int k = 0; k < this->_dimensions; k++) {
				if (k != 0) {
					out << " ";
				}
				out << this->_weights[calcIndex(i,j,k)];
			}
			out << std::endl;
		}
	}
}

std::fstream& SOM::GotoLine(std::fstream& file, unsigned int num){
    file.seekg(std::ios::beg);
    for(int i=0; i < num - 1; ++i){
        file.ignore(std::numeric_limits<std::streamsize>::max(),'\n');
    }
    return file;
}

void SOM::printDoubles(double *doubleList, unsigned int numDoubles, unsigned int numLines)
{
	unsigned int numPerLine = numDoubles/numLines;
	unsigned int counter = 0;
	while(counter < numDoubles)
	{
		for (int j = 0; j< numPerLine; j++)
		{
			std::cout << doubleList[counter] << " ";
			counter++;
		}
		std::cout << std::endl;
	}
}

//----------------------------------------------------
//	private SOM functions
//----------------------------------------------------

/*
	Load a trained SOM that was saved using the same algorithm as save_weights from an input stream
*/
void SOM::loadWeights(std::istream &in)
{
	// Load SOM dimensions first
	in >> this->_width >> this->_height;

	// Read first line of matrix to get the dimensionality of weights
	this->_dimensions = 0;
	std::string line;
	std::getline(in, line);
	std::getline(in, line);
	std::stringstream ss(line);
	std::vector<double> line1;
	double temp;
	while (ss >> temp) {
		this->_dimensions++;
		line1.push_back(temp);
	}

	// Put first line of matrix into an array in the 3d weights array
	this->_weights = new double[_width * _height * _dimensions];
	for (int k = 0; k < this->_dimensions; k++) {
		_weights[calcIndex(0,0,_dimensions - k - 1)] = line1.back();
		line1.pop_back();
	}

	// Read the rest of the 3d array in
	for (int i = 0; i < this->_width; i++) {
		for (int j = (i == 0 ? 1 : 0); j < this->_height; j++) {
			for (int k = 0; k < _dimensions; k++) {
				in >> this->_weights[calcIndex(i,j,k)];
			}
		}
	}
}

/*
	Normalizes given data to be between 0 and 1 for each feature
*/
void SOM::normalizeData(double *trainData)
{
	// Find the max and min value for each feature then use it to normalize the feature
	this->_featureMaxes = new double[this->_dimensions];
	this->_featureMins = new double[this->_dimensions];
	for (int d = 0; d < this->_dimensions; d++)
	{
		this->_featureMaxes[d] = -std::numeric_limits<double>::max();
		this->_featureMins[d] = std::numeric_limits<double>::max();
		for (int i = 0; i < this->_numExamples; i++)
		{
			if (trainData[i*this->_dimensions + d] > this->_featureMaxes[d]) {
				this->_featureMaxes[d] = trainData[i*_dimensions + d];
			}
			if (trainData[i*this->_dimensions + d] < this->_featureMins[d]) {
				this->_featureMins[d] = trainData[i*this->_dimensions + d];
			}
		}
		for (int i = 0; i < this->_numExamples; i++) {
			if ((this->_featureMaxes[d] - this->_featureMins[d]) <= std::numeric_limits<double>::min())
			{
				trainData[i*_dimensions + d] = 0;
			}
			else {
				trainData[i*_dimensions + d] = (trainData[i*_dimensions + d] - this->_featureMins[d])/(this->_featureMaxes[d]-this->_featureMins[d]);
			}
		}
	}
}

/*
	Calculate the index of a weight at node (x,y), dimension = d in the weights array
*/
int SOM::calcIndex(int x, int y, int d) {
	return (x*_height + y)*_dimensions + d;
}

void SOM::initMultiGPUSetup(int &ngpus) {
	
}

void SOM::initNumDenom(double *&numer, double *&denom) {
	numer = (double *)malloc(this->_mapSize * this->_dimensions * sizeof(double));
	denom = (double *)malloc(this->_mapSize * sizeof(double));
	for (int i = 0; i < this->_mapSize; i++) {
		denom[i] = 0.0;
		for (int j = 0; j < this->_dimensions; j++) {
			numer[i*this->_dimensions + j] = 0.0;
		}
	}
}

void SOM::initGPUTrainData(const int ngpus, double *trainData, double **d_train, int *GPU_EXAMPLES, int *GPU_OFFSET) {
	#pragma omp parallel
	{
		int NUM_BLOCKS;
		int NUM_THREADS = 256;
		int gpu = omp_get_thread_num();

		NUM_BLOCKS = (int)ceil((float)(GPU_EXAMPLES[gpu] * this->_dimensions)/NUM_THREADS);
		double *temp_d_train;

		gpuErrchk(hipSetDevice(gpu));
		gpuErrchk(hipMalloc(&temp_d_train, GPU_EXAMPLES[gpu] * this->_dimensions * sizeof(double)));
		gpuErrchk(hipMemcpy(temp_d_train, &trainData[GPU_OFFSET[gpu]], GPU_EXAMPLES[gpu] * this->_dimensions * sizeof(double), hipMemcpyHostToDevice));
		// Convert data from row major order to 
		rowToColumnMajor<<<NUM_BLOCKS, NUM_THREADS>>>(temp_d_train, d_train[gpu], GPU_EXAMPLES[gpu], this->_dimensions, GPU_EXAMPLES[gpu] * this->_dimensions);
		gpuErrchk(hipDeviceSynchronize());
		gpuErrchk(hipFree(temp_d_train));
	}
}

void SOM::initGPUTrainMemory(const int ngpus, hipblasHandle_t *&handles, double **&d_train, double **&d_weights, double **&d_numer, double **&d_denom, int *&GPU_EXAMPLES, int *&GPU_OFFSET, int num_examples) {
	handles = (hipblasHandle_t *)malloc(ngpus * sizeof(hipblasHandle_t));
	d_train = (double **)malloc(ngpus * sizeof(double *));
	d_weights = (double **)malloc(ngpus * sizeof(double *));
	d_numer = (double **)malloc(ngpus * sizeof(double *));
	d_denom = (double **)malloc(ngpus * sizeof(double *));
	GPU_EXAMPLES = (int *)malloc(ngpus * sizeof(int));
	GPU_OFFSET = (int *)malloc(ngpus * sizeof(int));
	GPU_OFFSET[0] = 0;

	for (int gpu = 0; gpu < ngpus; gpu++) {
		gpuErrchk(hipSetDevice(gpu));
		// Create cublas handles associated with each device
		hipblasCreate(&handles[gpu]);

		// Set the number of examples allocated to each GPU simply by equal division
		GPU_EXAMPLES[gpu] = num_examples/ngpus;
		if (gpu < ngpus-1)
			GPU_OFFSET[gpu+1] = GPU_OFFSET[gpu] + GPU_EXAMPLES[gpu];
		// Allocate remainder examples to last gpu
		else
			GPU_EXAMPLES[gpu] += num_examples - (GPU_OFFSET[gpu] + GPU_EXAMPLES[gpu]);
		
		// Allocate space for current GPU's share of the examples
		gpuErrchk(hipMalloc(&d_train[gpu], GPU_EXAMPLES[gpu] * this->_dimensions * sizeof(double)));
		// Allocate space for current GPU's copy of the map
		gpuErrchk(hipMalloc(&d_weights[gpu], this->_mapSize * this->_dimensions * sizeof(double)));
		// Allocate space for current GPU's copy of numerators and denominators
		gpuErrchk(hipMalloc(&d_numer[gpu], this->_mapSize * this->_dimensions * sizeof(double)));
		gpuErrchk(hipMalloc(&d_denom[gpu], this->_mapSize * sizeof(double)));
	}
}

void SOM::initGPUNumDenReducMem(const int ngpus, double **&gnumer, double **&gdenom) {
	gnumer = (double **)malloc(ngpus * sizeof(double *));
	gdenom = (double **)malloc(ngpus * sizeof(double *));
	for (int gpu = 0; gpu < ngpus; gpu++) {
		gnumer[gpu] = (double *)malloc(this->_mapSize * this->_dimensions * sizeof(double));
		gdenom[gpu] = (double *)malloc(this->_mapSize * sizeof(double));
	}
}

void SOM::initCodebook() {
	this->_weights = (double *)malloc(this->_mapSize * this->_dimensions * sizeof(double));
	for (int i = 0; i < this->_mapSize; i++) {
		for (int d = 0; d < this->_dimensions; d++) {
			this->_weights[i * this->_dimensions + d] = this->randWeight();
		}
	}
}

void SOM::initCodebookOnGPU(double **d_weights) {
	const int CODEBOOK_INIT_DEVICE = 0;
	gpuErrchk(hipSetDevice(CODEBOOK_INIT_DEVICE));
	hiprandGenerator_t gen;
	hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
	hipDeviceSynchronize();
	// TODO: hiprandSetPseudoRandomGeneratorSeed(gen, );
	hiprandGenerateUniformDouble(gen, d_weights[CODEBOOK_INIT_DEVICE], this->_mapSize * this->_dimensions);
	
	// Copy map from gpu to cpu
	this->_weights = (double *)malloc(this->_mapSize * this->_dimensions * sizeof(double));
	gpuErrchk(hipMemcpy(this->_weights, d_weights[CODEBOOK_INIT_DEVICE], this->_mapSize * this->_dimensions * sizeof(double), hipMemcpyDeviceToHost));
}

void SOM::setGPUCodebooks(double **d_weights) {
	#pragma omp parallel
	{
		int gpu = omp_get_thread_num();

		gpuErrchk(hipSetDevice(gpu));
		gpuErrchk(hipMemcpy(d_weights[gpu], this->_weights, this->_mapSize * this->_dimensions * sizeof(double), hipMemcpyHostToDevice));
	}
}

double SOM::randWeight()
{
	return (double)rand() / (RAND_MAX);
}